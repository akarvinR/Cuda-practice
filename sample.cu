#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <chrono>
#include <opencv2/opencv.hpp> 
const int n = 500'000'000;
template<typename T>
__global__ void vectorAdd(T* a, T* b, T* c, int numElements){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < numElements){
        c[i] = b[i] + a[i];
    }
}
template<typename T>
void vectorAddCPU(T *a, T *b ,T *c, int numElements){
    for(int i = 0;i<numElements;i++){
        c[i] = a[i] + b[i];
    }
}

bool check(float *c, float *a, float *b){
    for(int i = 0;i<n;i++){
        if(a[i] + b[i] != c[i]){
            std::cout << a[i] << " " << b[i] << " " << c[i] << std::endl;
            return false;
        }
    }
    return true;
}

void fill(float *a){
    for(int i = 0;i<n;i++){
        a[i] = rand()%1000000; 
    }
}
int main(){
    srand (time(NULL));
    
    cv::Mat image = cv::imread("./eagle.jpeg"); 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float* a = new float[n];
    float* b = new float[n];
    float *c = new float[n];
    float *rc = new float[n];
    float *d_a, *d_b, *d_c; 

    fill(a);
    fill(b);
    
    size_t size = sizeof(float)*n;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);


    hipEventRecord(start);

    int threadSize = 1024;
    int blockSize = n/threadSize + 1;
    vectorAdd<float><<<blockSize, threadSize>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if(check(c, a, b)){
        std::cout << "\nSucccesss in" << " " << milliseconds;
    }else{
        std::cout << "\n Failed in" << " " << milliseconds;
    }

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    vectorAddCPU<float>(rc, a, b, n);
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << " CPU TIME :" << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;   
}

